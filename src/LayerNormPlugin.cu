#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template<typename T>
__global__ void layerNormKernel(T *pInput, T *pOutput, float epsilon, const int N)
{
    const int tx = threadIdx.x;
    const int base_index = blockIdx.x * N;

    __shared__ T mean_shared, var_shared;

    // typedef hipcub::BlockReduce<T, 1024>            BlockReduce;
    // __shared__ typename BlockReduce::TempStorage temp;
    __shared__ T BlockReduce[1024];
    __shared__ uint BlockReduceN[1024];
    
    T _sum = 0;
    uint _ReduceN = 0;
    BlockReduceN[tx] = 0;
    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        _sum += v;
        _ReduceN++;
    }
    // T &ref0 = _sum;
    // T sum = BlockReduce(temp).Sum(ref0);
    BlockReduce[tx] = _sum / (T)_ReduceN;
    BlockReduceN[tx] = _ReduceN;
    __syncthreads();
    
    for (int stride = 512; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            uint rN = BlockReduceN[tx], lN = BlockReduceN[tx + stride];
            uint total = rN + lN;
            T rScale = rN > 0 ? (T)rN / (T)total : (T)0;
            T lScale = lN > 0 ? (T)lN / (T)total : (T)0;
            T r = rScale * BlockReduce[tx];
            T l = lScale * BlockReduce[tx + stride];
            BlockReduce[tx] = l + r;
            BlockReduceN[tx] = total;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        mean_shared = BlockReduce[0];
    __syncthreads();

    T _var_sum = 0;
    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        T moment = v - mean_shared, moment2 = moment * moment;
        _var_sum += moment2;
    }
    // T &ref1 = _var_sum;
    // T  var  = BlockReduce(temp).Sum(ref1);
    BlockReduce[tx] = _var_sum;
    __syncthreads();
    
    for (int stride = 512; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            BlockReduce[tx] += BlockReduce[tx + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0)
        var_shared = BlockReduce[0] / (T)N;
    __syncthreads();

    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        T moment = v - mean_shared;
        pOutput[base_index + tid] = moment * (T)rsqrtf(var_shared + (T)epsilon);
        // pOutput[base_index + tid] = (T)rsqrtf((T)epsilon); // 316.2278
        // pOutput[base_index + tid] = (T)epsilon; // 0
    }
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = inputDesc[0].dims.d[0], N = 1;
    for (int i = 1; i < inputDesc[0].dims.nbDims; ++i)
    {
        N *= inputDesc[0].dims.d[i];
    }
    int threadsPerBlock = 1024;
    if (inputDesc[0].type == DataType::kFLOAT)
    {
        layerNormKernel<float><<<nBlock, threadsPerBlock, 0, stream>>>((float*)inputs[0], (float*)outputs[0], epsilon_, N);
    }
    else
    {
        layerNormKernel<float><<<nBlock, threadsPerBlock, 0, stream>>>((float*)inputs[0], (float*)outputs[0], epsilon_, N);
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);