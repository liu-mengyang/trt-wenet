#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template<typename T>
__global__ void layerNormKernel(T *pInput, T *pOutput, float epsilon, const int N)
{
    const int base_index = blockIdx.x * N;

    __shared__ T mean_shared, var_shared;

    typedef hipcub::BlockReduce<T, 1024>            BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp;
    T _sum = 0;
    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        _sum += v;
    }
    T &ref0 = _sum;
    T sum = BlockReduce(temp).Sum(ref0);

    if (threadIdx.x == 0)
        mean_shared = sum / (T)N;
    __syncthreads();

    T _var_sum = 0;
    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        T moment = v - mean_shared, moment2 = moment * moment;
        _var_sum += moment2;
    }
    T &ref1 = _var_sum;
    T  var  = BlockReduce(temp).Sum(ref1);

    if (threadIdx.x == 0)
        var_shared = var / (T)N;
    __syncthreads();

    for (int tid=threadIdx.x; tid < N; tid += blockDim.x) {
        T v = pInput[base_index + tid];
        T moment = v - mean_shared;
        pOutput[base_index + tid] = moment * (T)rsqrtf(var_shared + (T)epsilon);
    }
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = inputDesc[0].dims.d[0], N = 1;
    for (int i = 1; i < inputDesc[0].dims.nbDims; ++i)
    {
        N *= inputDesc[0].dims.d[i];
    }
    int threadsPerBlock = 1024;
    if (inputDesc[0].type == DataType::kFLOAT)
    {
        layerNormKernel<float><<<nBlock, threadsPerBlock, 0, stream>>>((float*)inputs[0], (float*)outputs[0], epsilon_, N);
    }
    else
    {
        layerNormKernel<float><<<nBlock, threadsPerBlock, 0, stream>>>((float*)inputs[0], (float*)outputs[0], epsilon_, N);
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);