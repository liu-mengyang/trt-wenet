#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template<typename T>
__inline__ __device__ T Div(T a, T b);

template<>
__inline__ __device__ float Div<float>(float a, float b) {
  return a / b;
}

/* https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance
# For a new value newValue, compute the new count, new mean, the new M2.
# mean accumulates the mean of the entire dataset
# M2 aggregates the squared distance from the mean
# count aggregates the number of samples seen so far
def update(existingAggregate, newValue):
    (count, mean, M2) = existingAggregate
    count += 1
    delta = newValue - mean
    mean += delta / count
    delta2 = newValue - mean
    M2 += delta * delta2
    return (count, mean, M2)

# Retrieve the mean, variance and sample variance from an aggregate
def finalize(existingAggregate):
    (count, mean, M2) = existingAggregate
    if count < 2:
        return float("nan")
    else:
        (mean, variance, sampleVariance) = (mean, M2 / count, M2 / (count - 1))
        return (mean, variance, sampleVariance)
*/
template<typename T> // 迭代式Welford
inline __device__ void WelfordCombine(T val, T* mean, T* m2, T* count) {
  // Use Welford Online algorithem to compute mean and variance
  // For more details you can refer to:
  // https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance#Welford's_online_algorithm
  *count += 1;
  T delta1 = val - *mean;
  *mean += Div(delta1, *count);
  T delta2 = val - *mean;
  *m2 += delta1 * delta2;
}

/* https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance
def parallel_variance(n_a, avg_a, M2_a, n_b, avg_b, M2_b):
    n = n_a + n_b
    delta = avg_b - avg_a
    M2 = M2_a + M2_b + delta ** 2 * n_a * n_b / n
    var_ab = M2 / (n - 1)
    return var_ab
*/
template<typename T> // 并行式Welford
inline __device__ void WelfordCombine(T b_mean, T b_m2, T b_count, T* mean, T* m2, T* count) {
  if (b_count == 0) { return; }
  T new_count = *count + b_count;
  T delta = b_mean - *mean;
  T nb_over_n = Div(b_count, new_count);
  *mean += delta * nb_over_n;
  *m2 += b_m2 + delta * delta * (*count) * nb_over_n;
  *count = new_count;
}

template<typename T> // 并行Reduce式Welford
__inline__ __device__ void WelfordWarpReduce(T thread_mean, T thread_m2, T thread_count, T* mean, T* m2, T* count) {
  *mean = thread_mean;
  *m2 = thread_m2;
  *count = thread_count;
  for (int delta = 32 / 2; delta > 0; delta /= 2) { // 一次次Reduce
    // 获取高位Thread中的mean m2 count
    // 应该还是有很多“空转”，这里明显第一次计算是前16个Thread获取后16个Thread的数据计算，而后16个Thread中的数据就没用了
    T b_mean = __shfl_down_sync(0xffffffff, *mean, delta, 32);
    T b_m2 = __shfl_down_sync(0xffffffff, *m2, delta, 32);
    T b_count = __shfl_down_sync(0xffffffff, *count, delta, 32);
    // 执行计算
    WelfordCombine(b_mean, b_m2, b_count, mean, m2, count);
  }
}

template<typename T> // 并行Reduce式Welford，算完取数据
__inline__ __device__ void WelfordWarpAllReduce(T thread_mean, T thread_m2, T thread_count, T* mean, T* m2, T* count) {
  WelfordWarpReduce<T>(thread_mean, thread_m2, thread_count, mean, m2, count);
  // 最后就是从Warp的第一个Thread里面取数据
  *mean = __shfl_sync(0xffffffff, *mean, 0, 32);
  *m2 = __shfl_sync(0xffffffff, *m2, 0, 32);
  *count = __shfl_sync(0xffffffff, *count, 0, 32);
}

template<typename T>
__global__ void layerNormKernel(T *pInput, T *pOutput, float epsilon, const int N)
{
    const int base_index = blockIdx.x * N;

    T mean = 0, m2 = 0, count = 0;
    for (int i=threadIdx.x; i < N; i += blockDim.x) {
        T v = pInput[base_index + i];
        WelfordCombine<T>(v, &mean, &m2, &count); // 先1024个Thread并行执行顺序式的Welford，算出1024个结果
    }
    __syncthreads();

    WelfordWarpAllReduce<T>(mean, m2, count, &mean, &m2, &count); // 对这1024个Thread执行Reduce式的Welford

    __shared__ T s_mean[32], s_m2[32], s_count[32]; // 1024个进程，每32为一Warp开算一轮Reduce，共产生1024/32=32个结果

    if (threadIdx.x % 32 == 0) { // 这32个结果分布在第0,32,64,...号Thread上
        uint i = threadIdx.x / 32; // 将这32个结果收集起来
        s_mean[i] = mean;
        s_m2[i] = m2;
        s_count[i] = count;
    }
    __syncthreads();

    if (threadIdx.x < 32) { // 然后再用一个Warp开算一轮Reduce
        mean = s_mean[threadIdx.x];
        m2 = s_m2[threadIdx.x];
        count = s_count[threadIdx.x];
        WelfordWarpAllReduce<T>(mean, m2, count, &mean, &m2, &count);
    }
    __syncthreads();

    __shared__ T mean_shared, var_shared;
    if (threadIdx.x == 0) { // 那么此时，0号Thread上的就是最终结果了
        mean_shared = mean;
        var_shared = Div(m2, count);
    }
    __syncthreads();
    mean = mean_shared;
    T var = var_shared;
    
    for (int i=threadIdx.x; i < N; i += blockDim.x) {
        T v = pInput[base_index + i];
        pOutput[base_index + i] = (v - mean) * (T)rsqrtf(var + (T)epsilon);
    }
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1], N = 1;
    for (int i = 2; i < inputDesc[0].dims.nbDims; ++i)
    {
        N *= inputDesc[0].dims.d[i];
    }
    if (inputDesc[0].type == DataType::kFLOAT)
    {
        layerNormKernel<float><<<nBlock, 1024, 0, stream>>>((float*)inputs[0], (float*)outputs[0], epsilon_, N);
    }
    else if (inputDesc[0].type == DataType::kHALF)
    {
        layerNormKernel<__half><<<nBlock, 1024, 0, stream>>>((__half*)inputs[0], (__half*)outputs[0], epsilon_, N);
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);