#include "hip/hip_runtime.h"
#include "LayerNormPlugin.h"

using namespace nvinfer1;

PluginFieldCollection    LayerNormPluginCreator::fc_ {};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template<typename T, int n>
__global__ void layerNormKernel(T *pInput, T *pOutput, float epsilon)
{
    const int tx = threadIdx.x, index = blockIdx.x * n + threadIdx.x;

    T _x = pInput[index];

    __shared__ T mean_shared, var_shared;

    typedef hipcub::BlockReduce<T, n>               BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp;
    T &                                          ref0 = _x;
    T                                            sum  = BlockReduce(temp).Sum(ref0);
    //__syncthreads();
    if (tx == 0)
        mean_shared = sum / (T)n;
    __syncthreads();

    T  moment = _x - mean_shared, moment2 = moment * moment;
    T &ref1 = moment2;
    T  var  = BlockReduce(temp).Sum(ref1);
    //__syncthreads();
    if (tx == 0)
        var_shared = var / (T)n;
    __syncthreads();

    pOutput[index] = moment * (T)rsqrtf(var_shared + (T)epsilon);
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc *inputDesc, const PluginTensorDesc *outputDesc, const void *const *inputs, void *const *outputs, void *workspace, hipStream_t stream) noexcept
{
    WHERE_AM_I();
    int nBlock = inputDesc[0].dims.d[0], nValuePerBlock = 1;
    for (int i = 1; i < inputDesc[0].dims.nbDims; ++i)
    {
        nValuePerBlock *= inputDesc[0].dims.d[i];
    }

    if (inputDesc[0].type == DataType::kFLOAT)
    {
        switch (nValuePerBlock)
        {
        //case 60: // 仅演示 cIn*hIn*wIn==60 的情况
        //    (layerNormKernel<float, 60>)<<<nBlock, nValuePerBlock, 0, stream>>>((float *)inputs[0], (float *)outputs[0], epsilon_);
        //    break;
        default: // shoulf NOT be here
            (layerNormKernel<float, 256>)<<<nBlock, nValuePerBlock, 0, stream>>>((float *)inputs[0], (float *)outputs[0], epsilon_);
            //printf("[LayerNormPlugin::enqueue] nValuePerBlock = %d is not supported\n", nValuePerBlock);
            break;
        }
    }
    else
    {
        switch (nValuePerBlock)
        {
        //case 60: // 仅演示 cIn*hIn*wIn==60 的情况
        //    (layerNormKernel<float, 60>)<<<nBlock, nValuePerBlock, 0, stream>>>((float *)inputs[0], (float *)outputs[0], epsilon_);
        //    break;
        default: // shoulf NOT be here
            (layerNormKernel<float, 256>)<<<nBlock, nValuePerBlock, 0, stream>>>((float *)inputs[0], (float *)outputs[0], epsilon_);
            //printf("[LayerNormPlugin::enqueue] nValuePerBlock = %d is not supported\n", nValuePerBlock);
            break;
        }
    }
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);